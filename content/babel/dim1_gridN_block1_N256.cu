#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#define N 512
__global__ void add(int *a, int *b, int *c){
  int tid = threadIdx.x; // handle the data at this index
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}
int main()
{
  int a[N], b[N], c[N], i;
  int *dev_a, *dev_b, *dev_c;
  hipMalloc((void**)&dev_c, N*sizeof(int));
  hipMalloc((void**)&dev_b, N*sizeof(int));
  hipMalloc((void**)&dev_a, N*sizeof(int));
  for(i=0; i < N; i++)
    {
      a[i] = -i;
      b[i] = i*i*i;
    }
  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
  add <<<1, N>>>(dev_a, dev_b, dev_c);
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
  for(i=0; i < N; i++)
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  hipFree(dev_c);
  hipFree(dev_b);
  hipFree(dev_a);
  return 0;
}
