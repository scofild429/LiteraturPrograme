#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
  int index = threadIdx.x;
  int stride = blockDim.x;

  for(int i = index; i < n; i += stride){
    out[i] = a[i] + b[i];
  }
}

int main(){
  float *a, *b, *out;
  float *d_a, *d_b, *d_out; 

  // Allocate host memory
  a   = (float*)malloc(sizeof(float) * N);
  b   = (float*)malloc(sizeof(float) * N);
  out = (float*)malloc(sizeof(float) * N);

  // Initialize host arrays
  for(int i = 0; i < N; i++){
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  // Allocate device memory 
  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMalloc((void**)&d_out, sizeof(float) * N);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  // Executing kernel 
  vector_add<<<1,256>>>(d_out, d_a, d_b, N);

  // Transfer data back to host memory
  hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);



  // Verification
  for(int i = 0; i < N; i++){
    assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
  }

  printf("PASSED with %f ms\n", elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);


  // Deallocate device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  // Deallocate host memory
  free(a); 
  free(b); 
  free(out);
}
