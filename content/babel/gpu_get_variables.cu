#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <hiprand/hiprand_kernel.h>
#define N 32
__global__ void add(
                    int *dthreadIdx_x,
                    int *dthreadIdx_y,
                    int *dthreadIdx_z,
                    int *dblockIdx_x,
                    int *dblockIdx_y,
                    int *dblockIdx_z,
                    int *dblockDim_x,
                    int *dblockDim_y,
                    int *dblockDim_z,
                    int *dgridDim_x,
                    int *dgridDim_y,
                    int *dgridDim_z){
  *dthreadIdx_x = threadIdx.x;
  *dthreadIdx_y = threadIdx.y;
  *dthreadIdx_z = threadIdx.z;
  *dblockIdx_x = blockIdx.x;
  *dblockIdx_y = blockIdx.y;
  *dblockIdx_z = blockIdx.z;
  *dblockDim_x = blockDim.x;
  *dblockDim_y = blockDim.y;
  *dblockDim_z = blockDim.z;
  *dgridDim_x = gridDim.x;
  *dgridDim_y = gridDim.y;
  *dgridDim_z = gridDim.z;
}
int main()
{
  int threadIdx_x = 0;
  int threadIdx_y = 0;
  int threadIdx_z = 0;
  int blockIdx_x = 0;
  int blockIdx_y = 0;
  int blockIdx_z = 0;
  int blockDim_x = 0;
  int blockDim_y = 0;
  int blockDim_z = 0;
  int gridDim_x = 0;
  int gridDim_y = 0;
  int gridDim_z = 0;

  int *dthreadIdx_x;
  int *dthreadIdx_y;
  int *dthreadIdx_z;
  int *dblockIdx_x;
  int *dblockIdx_y;
  int *dblockIdx_z;
  int *dblockDim_x;
  int *dblockDim_y;
  int *dblockDim_z;
  int *dgridDim_x;
  int *dgridDim_y;
  int *dgridDim_z;

  hipMalloc((void **)&dthreadIdx_x, sizeof(int));
  hipMalloc((void **)&dthreadIdx_y, sizeof(int));
  hipMalloc((void **)&dthreadIdx_z, sizeof(int));
  hipMalloc((void **)&dblockIdx_x, sizeof(int));
  hipMalloc((void **)&dblockIdx_y, sizeof(int));
  hipMalloc((void **)&dblockIdx_z, sizeof(int));
  hipMalloc((void **)&dblockDim_x, sizeof(int));
  hipMalloc((void **)&dblockDim_y, sizeof(int));
  hipMalloc((void **)&dblockDim_z, sizeof(int));
  hipMalloc((void **)&dgridDim_x, sizeof(int));
  hipMalloc((void **)&dgridDim_y, sizeof(int));
  hipMalloc((void **)&dgridDim_z, sizeof(int));

  dim3 BlockPerGrid(1, 1, 1);
  dim3 ThreadsPerBlock(N, N, 1);
  add <<< BlockPerGrid, ThreadsPerBlock >>>(
                                            dthreadIdx_x,
                                            dthreadIdx_y,
                                            dthreadIdx_z,
                                            dblockIdx_x,
                                            dblockIdx_y,
                                            dblockIdx_z,
                                            dblockDim_x,
                                            dblockDim_y,
                                            dblockDim_z,
                                            dgridDim_x,
                                            dgridDim_y,
                                            dgridDim_z
                                            );
  hipMemcpy(&threadIdx_x,dthreadIdx_x, sizeof(int), hipMemcpyDeviceToHost);
  printf("threadIdx_x is %d \n", threadIdx_x);
  hipMemcpy(&threadIdx_y,dthreadIdx_y, sizeof(int), hipMemcpyDeviceToHost);
  printf("threadIdx_y is %d \n", threadIdx_y);
  hipMemcpy(&threadIdx_z,dthreadIdx_z, sizeof(int), hipMemcpyDeviceToHost);
  printf("threadIdx_z is %d \n", threadIdx_z);

  hipMemcpy(&blockIdx_x, dblockIdx_x, sizeof(int), hipMemcpyDeviceToHost);
  printf("blockIdx_x is %d \n", blockIdx_x);
  hipMemcpy(&blockIdx_y, dblockIdx_y, sizeof(int), hipMemcpyDeviceToHost);
  printf("blockIdx_y is %d \n", blockIdx_y);
  hipMemcpy(&blockIdx_z, dblockIdx_z, sizeof(int), hipMemcpyDeviceToHost);
  printf("blockIdx_z is %d \n", blockIdx_z);


  hipMemcpy(&blockDim_x, dblockDim_x, sizeof(int), hipMemcpyDeviceToHost);
  printf("blockDim_x is %d \n", blockDim_x);
  hipMemcpy(&blockDim_y, dblockDim_y, sizeof(int), hipMemcpyDeviceToHost);
  printf("blockDim_y is %d \n", blockDim_y);
  hipMemcpy(&blockDim_z, dblockDim_z, sizeof(int), hipMemcpyDeviceToHost);
  printf("blockDim_z is %d \n", blockDim_z);

  hipMemcpy(&gridDim_x, dgridDim_x, sizeof(int), hipMemcpyDeviceToHost);
  printf("gridDim_x is %d \n", gridDim_x);
  hipMemcpy(&gridDim_y, dgridDim_y, sizeof(int), hipMemcpyDeviceToHost);
  printf("gridDim_y is %d \n", gridDim_y);
  hipMemcpy(&gridDim_z, dgridDim_z, sizeof(int), hipMemcpyDeviceToHost);
  printf("gridDim_z is %d \n", gridDim_z);

  return 0;
}
