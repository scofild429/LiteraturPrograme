#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <hiprand/hiprand_kernel.h>
#define n 32
#define N n*n
__global__ void add(float *a, float *b, float *c){
  __shared__ float A[n][n];
  __shared__ float B[n][n];
  __shared__ float C[n][n];
  int tidx = threadIdx.x;
  int tidy = threadIdx.y;

  A[tidy][tidx] = a[tidx +n*tidy];
  B[tidy][tidx] = b[tidx +n*tidy];

  C[tidy][tidx]= A[tidy][tidx] + B[tidy][tidx];

  c[tidx +n*tidy] = C[tidy][tidx];
}
int main()
{
  float a[N], b[N], out[N];
  float *d_a, *d_b, *d_out; 

  for(int i = 0; i < n; i++){
    for (int j = 0; j < n; j++){
      a[i+j*n] = 1.0f + i;
      b[i+j*n] = 1.0f + i;
    }
  }

  // Allocate device memory 
  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMalloc((void**)&d_out, sizeof(float) * N);

  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  dim3 BlockPerGrid(1, 1, 1);
  dim3 ThreadsPerBlock(n, n, 1);
  add <<< BlockPerGrid, ThreadsPerBlock >>>(d_a, d_b, d_out);
  hipMemcpy(out, d_out, sizeof(int) * N, hipMemcpyDeviceToHost);
  for(int i=0; i < N; i++){
    printf("a[%d] + b[%d] == out[%d] is %f + %f = %f\n", i,i,i, a[i], b[i], out[i]);
  }
  printf("Passed\n");
  hipFree(d_out);
  hipFree(d_b);
  hipFree(d_a);
  return 0;
}
