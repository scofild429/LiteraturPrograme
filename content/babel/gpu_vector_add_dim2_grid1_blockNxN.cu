#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <hiprand/hiprand_kernel.h>
#define N 512
__global__ void add(int *a, int *b, int *c){
  int tid = threadIdx.x + threadIdx.y*blockDim.x + blockDim.x*blockDim.y*blockIdx.x; 
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}
int main()
{
  int a[N], b[N], c[N], i;
  int *dev_a, *dev_b, *dev_c;
  hipMalloc((void**)&dev_c, N*sizeof(int));
  hipMalloc((void**)&dev_b, N*sizeof(int));
  hipMalloc((void**)&dev_a, N*sizeof(int));
  for(i=0; i < N; i++)
    {
      a[i] = 1;
      b[i] = 2;
    }
  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
  dim3 BlockPerGrid(1, 1, 1);
  dim3 ThreadsPerBlock(128, 4, 1);
  add <<< BlockPerGrid, ThreadsPerBlock >>>(dev_a, dev_b, dev_c);
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
  for(i=0; i < N; i++){
    assert(a[i] + b[i] == c[i]);
  }
  printf("Passed\n");
  hipFree(dev_c);
  hipFree(dev_b);
  hipFree(dev_a);
  return 0;
}
