#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define N 512
__global__ void add(int *a, int *b, int *c){
  int tid = threadIdx.x + threadIdx.y*blockDim.x + blockDim.x*blockDim.y*blockIdx.x; 
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}
int main()
{
  float a[N][N], b[N][N], out[N][N];
  float d_a[N][N], d_b[N][N], d_out[N][N]; 

  for(int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      a[i][j] = 1.0f;
      b[i][j] = 2.0f;
    }
  }

  // Allocate device memory 
  hipMalloc((void**)&d_a, sizeof(float) * N *N);
  hipMalloc((void**)&d_b, sizeof(float) * N * N);
  hipMalloc((void**)&d_out, sizeof(float) * N * N);

  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N * N, hipMemcpyHostToDevice);

  dim3 BlockPerGrid(1, 1, 1);
  dim3 ThreadsPerBlock(128, 4, 1);
  add <<< BlockPerGrid, ThreadsPerBlock >>>(dev_a, dev_b, dev_c);
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
  for(i=0; i < N; i++){
    assert(a[i] + b[i] == c[i]);
  }
  printf("Passed\n");
  hipFree(dev_c);
  hipFree(dev_b);
  hipFree(dev_a);
  return 0;
}
